#include <stdio.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
//#include "helper_functions.h"

#define MATRIX_1D

#define cudaCheckErrors() {                                                               \
        hipError_t error=hipGetLastError();                                                        \
        if(error!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));        \
            exit(EXIT_FAILURE);                                                                      \
        }                                                                                            \
    }


//////////////////////////////////////////////////////////////////////////////////////
//Host Code
//////////////////////////////////////////////////////////////////////////////////////
void convolutionHostRow(double **input, double **output, int dim_x, int dim_y) {

	for (int x= 0; x < dim_x; x++) {
		for (int y=0 ; y < dim_y; y++) {

			output[x][y] = -input[x][y];
		}
	}
}
void convolutionHostColumn(double **input, double **output, int dim_x, int dim_y) {

	for (int x= 0; x < dim_x; x++) {
		for (int y=0 ; y < dim_y; y++) {

			output[x][y] = input[y][x];
		}
	}
}

//////////////////////////////////////////////////////////////////////////////////////
//Device Code
//////////////////////////////////////////////////////////////////////////////////////
__global__ void
convolutionDeviceRow(double *input, double *output, int dim_x, int dim_y){

	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int y = blockIdx.x * blockDim.x + threadIdx.x;

	int pos_x = x + y*dim_x;

	output[pos_x] = -input[pos_x];

}

__global__ void
convolutionDeviceColumn(double *input, double *output, int dim_x, int dim_y){

	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int y = blockIdx.x * blockDim.x + threadIdx.x;

	output[x + y*blockDim.x] = input[y + x*blockDim.y];
}

int main(int argc, char const *argv[]) {

	double
	**h_matrix ,
	**h_buffer ,
	**h_OutputCPU ,
	*h_OutputGPU ,
	*h_trans_matrix;

	double
	*d_matrix ,
	*d_buffer ,
	*d_output;

	int dim_x = 1, dim_y = 1, max_withd = 3;

	if ( argc != 3) {
		printf("Insuffisend arguments exiting\n");
	} else {
		dim_x = atoi(argv[1]);
		dim_y = atoi(argv[2]);
	}

	printf("Initializing host matricies...\n");
	h_matrix = (double**)malloc(sizeof(double*)*dim_y);
	if ( h_matrix == NULL){
		fprintf(stderr, "Error in Host Matrix allocation\n");
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_x; i++){
		h_matrix[i] = (double*)realloc(h_matrix[i], sizeof(double) * dim_y);
		if (h_matrix[i] == NULL){
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
		for ( int j = 0; j < dim_y; j++) {
			h_matrix[i][j] = j + (i * dim_x);
		}
	}

	h_buffer = (double**)malloc(sizeof(double*)*dim_y);
	if ( h_buffer == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_x; i++){
		h_buffer[i] = (double*)realloc(h_buffer[i], sizeof(double) * dim_y);
		if ( h_buffer[i] == NULL) {
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
	}

	h_OutputCPU = (double**)malloc(sizeof(double*)*dim_y);
	if ( h_buffer == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_x; i++){
		h_OutputCPU[i] = (double*)realloc(h_OutputCPU[i], sizeof(double) * dim_y);
		if ( h_OutputCPU[i] == NULL) {
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
	}

	h_OutputGPU = (double*)malloc(sizeof(double)*dim_y*dim_x);
	if ( h_buffer == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}

	h_trans_matrix = (double*)malloc(sizeof(double)*dim_x*dim_y);
	if (h_trans_matrix == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}

	printf("Initializing device matricies...\n");
	d_matrix = NULL;
	hipMalloc((void**)&d_matrix, sizeof(double)*dim_x*dim_y);

	for ( int i =0 ; i < dim_x; i++) {
		for ( int j = 0; j < dim_y; j++) {
			h_trans_matrix[j + (i * dim_x)] = h_matrix[i][j];
		}
	}

	d_buffer = NULL;
	hipMalloc((void**)&d_buffer, sizeof(double)*dim_x*dim_y);

	d_output = NULL;
	hipMalloc((void**)&d_output, sizeof(double)*dim_x*dim_y);
#ifdef MATRIX_2D
	for ( int i = 0; i < dim_y; i++) {
		//d_output[i] = NULL;
		cudaMalloc((void**)&d_output[i], sizeof(double)*dim_x);
		cudaCheckErrors();
	}
#endif

	printf("Transfering data to cuda Device...\n");

	hipMemcpy( d_matrix, h_trans_matrix, dim_x*dim_y*sizeof(double) , hipMemcpyHostToDevice);
	cudaCheckErrors();

	printf("Running CPU code...\n");

	convolutionHostRow(h_matrix, h_buffer, dim_x, dim_y);
	convolutionHostColumn(h_buffer, h_OutputCPU, dim_x, dim_y);

	//kernel prep
	printf("Running GPU code...\n");

	int threadsPerBlock;
	int blocksPerGrid;
	if ( dim_x > 32 || dim_y > 32) {
		threadsPerBlock = 32;
		blocksPerGrid = dim_x/32;
	} else {
		threadsPerBlock = dim_x;
		blocksPerGrid = 1;
	}
	dim3 threads(threadsPerBlock, threadsPerBlock);
	dim3 grid(blocksPerGrid, blocksPerGrid);

	printf("CUDA kernel launch %d blocks of %d threads\n", grid.x*grid.y, threads.x*threads.y);

	convolutionDeviceRow<<<grid, threads>>>( d_matrix, d_buffer, dim_x, dim_y);
	hipDeviceSynchronize();
	cudaCheckErrors();

	printf("CUDA kernel launch %d blocks of %d threads\n", grid.x*grid.y, threads.x*threads.y);
	convolutionDeviceColumn<<<grid, threads>>>( d_buffer, d_output, dim_x, dim_y);
	hipDeviceSynchronize();
	cudaCheckErrors();

	hipMemcpy( h_OutputGPU, d_output, sizeof(double)*dim_x*dim_y, hipMemcpyDeviceToHost);
	cudaCheckErrors();

	printf("\nInput Matrix\n********************************************************************\n" );
	for ( int i = 0; i < dim_x; i++){
		for (int j = 0; j < dim_y; j++) {
			printf(" %*g", max_withd, h_matrix[i][j]);
		}
		printf("\n");
	}

	printf("\nCPU Matrix\n********************************************************************\n" );
	for ( int i = 0; i < dim_x; i++){
		for (int j = 0; j < dim_y; j++) {
			printf(" %*g", max_withd, h_OutputCPU[i][j]);
		}
		printf("\n");
	}

	printf("\nGPU Matrix\n********************************************************************\n" );
	for ( int i = 0; i < dim_x; i++){
		for (int j = 0; j < dim_y; j++) {
			printf(" %*g", max_withd, h_OutputGPU[j + (i * dim_x)]);
		}
		printf("\n");
	}

	for ( int i = 0; i < dim_x; i++) {
			free(h_OutputCPU[i]);
			free(h_buffer[i]);
			free(h_matrix[i]);
	}
	free(h_trans_matrix);
	free(h_OutputGPU);

	hipFree(d_output);
	cudaCheckErrors();

	hipFree(d_matrix);
	cudaCheckErrors();

	hipFree(d_buffer);
	cudaCheckErrors();


	hipDeviceReset();
	cudaCheckErrors();


	return 0;
}
