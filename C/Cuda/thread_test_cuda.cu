#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "gputimer.h"
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Utilities and system includes
//#include "helper_functions.h"

#define DEBUG

#define cudaCheckErrors() {                                                               \
        hipError_t error=hipGetLastError();                                                        \
        if(error!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));        \
            exit(EXIT_FAILURE);                                                                      \
        }                                                                                            \
    }


//////////////////////////////////////////////////////////////////////////////////////
//Host Code
//////////////////////////////////////////////////////////////////////////////////////
void convolutionHostRow(double **input, double **output, int dim_x, int dim_y) {

	for (int x= 0; x < dim_x; x++) {
		for (int y=0 ; y < dim_y; y++) {

			output[x][y] = -input[x][y];
		}
	}
}
void convolutionHostColumn(double **input, double **output, int dim_x, int dim_y) {

	for (int x= 0; x < dim_x; x++) {
		for (int y=0 ; y < dim_y; y++) {

			output[x][y] = input[y][x];
		}
	}
}

//////////////////////////////////////////////////////////////////////////////////////
//Device Code
//////////////////////////////////////////////////////////////////////////////////////
__global__ void
convolutionDeviceRow(double *input, double *output, int dim_x, int dim_y){

	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int y = blockIdx.x * blockDim.x + threadIdx.x;

	int pos_x = x + y*dim_x;

	output[pos_x] = -input[pos_x];

}

__global__ void
convolutionDeviceColumn(double *input, double *output, int dim_x, int dim_y){

	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int y = blockIdx.x * blockDim.x + threadIdx.x;

	output[x + y*blockDim.x] = input[y + x*blockDim.y];
}

int main(int argc, char const *argv[]) {

	double
	**h_matrix ,
	**h_buffer ,
	**h_OutputCPU ,
	*h_OutputGPU ,
	*h_trans_matrix;

	double
	*d_matrix ,
	*d_buffer ,
	*d_output;

	double overal_GPU_time = 0, overal_CPU_time = 0, overal_data_transfer_time = 0;
	clock_t start, stop;
	GpuTimer timer;

	int dim_x = 1, dim_y = 1;
#ifdef DEBUG
	int max_withd = 3;
#endif

	if ( argc != 3) {
		printf("Insuffisend arguments exiting\n");
		exit(EXIT_FAILURE);
	} else {
		dim_x = atoi(argv[1]);
		dim_y = atoi(argv[2]);
	}

	printf("Initializing host matricies...\n");
	h_matrix = (double**)malloc(sizeof(double*)*dim_x);
	if ( h_matrix == NULL){
		fprintf(stderr, "Error in Host Matrix allocation\n");
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_y; i++){
		h_matrix[i] = (double*)malloc( sizeof(double) * dim_y);
		if (h_matrix[i] == NULL){
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
		for ( int j = 0; j < dim_y; j++) {
			h_matrix[i][j] = j + (i * dim_x);
		}
	}

	h_buffer = (double**)malloc(sizeof(double*)*dim_y);
	if ( h_buffer == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_x; i++){
		h_buffer[i] = (double*)realloc(h_buffer[i], sizeof(double) * dim_y);
		if ( h_buffer[i] == NULL) {
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
	}

	h_OutputCPU = (double**)malloc(sizeof(double*)*dim_y);
	if ( h_buffer == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_x; i++){
		h_OutputCPU[i] = (double*)realloc(h_OutputCPU[i], sizeof(double) * dim_y);
		if ( h_OutputCPU[i] == NULL) {
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
	}

	h_OutputGPU = (double*)malloc(sizeof(double)*dim_y*dim_x);
	if ( h_buffer == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}

	h_trans_matrix = (double*)malloc(sizeof(double)*dim_x*dim_y);
	if (h_trans_matrix == NULL) {
		fprintf(stderr, "Error in Host Matrix allocation\n" );
		exit(EXIT_FAILURE);
	}

	printf("Initializing device matricies...\n");
	d_matrix = NULL;
	hipMalloc((void**)&d_matrix, sizeof(double)*dim_x*dim_y);

	for ( int i =0 ; i < dim_x; i++) {
		for ( int j = 0; j < dim_y; j++) {
			h_trans_matrix[j + (i * dim_x)] = h_matrix[i][j];
		}
	}

	d_buffer = NULL;
	hipMalloc((void**)&d_buffer, sizeof(double)*dim_x*dim_y);

	d_output = NULL;
	hipMalloc((void**)&d_output, sizeof(double)*dim_x*dim_y);

	printf("Transfering data to cuda Device...\n");

	timer.Start();
	hipMemcpy( d_matrix, h_trans_matrix, dim_x*dim_y*sizeof(double) , hipMemcpyHostToDevice);
	timer.Stop();
	overal_data_transfer_time += timer.Elapsed();
	cudaCheckErrors();

	printf("Running CPU code...\n");

	start = clock();
	convolutionHostRow(h_matrix, h_buffer, dim_x, dim_y);
	convolutionHostColumn(h_buffer, h_OutputCPU, dim_x, dim_y);
	stop = clock();

	//kernel prep
	printf("Running GPU code...\n");

	int threadsPerBlock;
	int blocksPerGrid;
	if ( dim_x > 32 || dim_y > 32) {
		threadsPerBlock = 32;
		blocksPerGrid = dim_x/32;
	} else {
		threadsPerBlock = dim_x;
		blocksPerGrid = 1;
	}
	dim3 threads(threadsPerBlock, threadsPerBlock);
	dim3 grid(blocksPerGrid, blocksPerGrid);

	printf("CUDA kernel launch %d blocks of %d threads\n", grid.x*grid.y, threads.x*threads.y);

	timer.Start();
	convolutionDeviceRow<<<grid, threads>>>( d_matrix, d_buffer, dim_x, dim_y);
	timer.Stop();
	overal_GPU_time += timer.Elapsed();
	hipDeviceSynchronize();
	cudaCheckErrors();

	printf("CUDA kernel launch %d blocks of %d threads\n", grid.x*grid.y, threads.x*threads.y);

	timer.Start();
	convolutionDeviceColumn<<<grid, threads>>>( d_buffer, d_output, dim_x, dim_y);
	timer.Stop();
	overal_GPU_time += timer.Elapsed();

	hipDeviceSynchronize();
	cudaCheckErrors();

	timer.Start();
	hipMemcpy( h_OutputGPU, d_output, sizeof(double)*dim_x*dim_y, hipMemcpyDeviceToHost);
	timer.Stop();
	overal_data_transfer_time += timer.Elapsed();

	cudaCheckErrors();

	printf("\nTime elapsed on GPU( computation) = %g ms\n", overal_GPU_time);
	printf("Time elapsed on GPU( memory transfers) = %g ms\n", overal_data_transfer_time);
	printf("\nTime elapsed on GPU( overal) = %g ms\n", overal_GPU_time + overal_data_transfer_time);



	overal_CPU_time = (double)(stop - start) * 1000.0 / CLOCKS_PER_SEC ;
	printf ("\nTime elapsed on CPU = %g ms\n", overal_CPU_time);
#ifdef DEBUG

	printf("\nInput Matrix\n********************************************************************\n" );
	for ( int i = 0; i < dim_x; i++){
		for (int j = 0; j < dim_y; j++) {
			printf(" %*g", max_withd, h_matrix[i][j]);

		}
		printf("\n");
	}

	printf("\nCPU Matrix\n********************************************************************\n" );
	for ( int i = 0; i < dim_x; i++){
		for (int j = 0; j < dim_y; j++) {
			printf(" %*g", max_withd, h_OutputCPU[i][j]);
		}
		printf("\n");
	}

	printf("\nGPU Matrix\n********************************************************************\n" );
	for ( int i = 0; i < dim_x; i++){
		for (int j = 0; j < dim_y; j++) {
			printf(" %*g", max_withd, h_OutputGPU[j + (i * dim_x)]);
		}
		printf("\n");
	}
#endif


	for ( int i = 0; i < dim_x; i++) {
			free(h_OutputCPU[i]);
			free(h_buffer[i]);
			free(h_matrix[i]);
	}
	free(h_trans_matrix);
	free(h_OutputGPU);

	hipFree(d_output);
	cudaCheckErrors();

	hipFree(d_matrix);
	cudaCheckErrors();

	hipFree(d_buffer);
	cudaCheckErrors();


	hipDeviceReset();
	cudaCheckErrors();


	return 0;
}
