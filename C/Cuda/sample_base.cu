
#include <hip/hip_runtime.h>
#include<stdio.h>

//Compile with nvcc -arch sm_20 -lineinfo -Xcompiler -Wall -O4 -g sample_base.cu -o hello
//Check cuda call for errors, Call after each cuda call
#define cudaCheckError() {                                                                   \
	hipError_t e=hipGetLastError();                                                        \
	if(e!=hipSuccess) {                                                                     \
		printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
		exit(EXIT_FAILURE);                                                                  \
	}                                                                                        \
}


//This is the kernel that runs on the GPU
__global__ void hello_from_GPU()
{
	//printf("Hello world from the device! block=%d, thread=%d\n", blockIdx.x, threadIdx.x);
}

//This is the main function
int main(int argc, char **argv)
{
	double **hostMatrix, *hostGpuOutput;
	double *deviceMatrix, *deviceMatrixTransposed;


	int dim_x = 1, dim_y = 1;
#ifdef DEBUG
	int max_withd = 3;
#endif

	if ( argc != 3)
	{
		printf("Insuffisend arguments exiting\n");
		exit(EXIT_FAILURE);
	}
	else
	{
		dim_x = atoi(argv[1]);
		dim_y = atoi(argv[2]);
	}
	hostMatrix = (double**)malloc(sizeof(double*)*dim_x);
	if ( hostMatrix == NULL){
		fprintf(stderr, "Error in Host Matrix allocation\n");
		exit(EXIT_FAILURE);
	}
	for ( int i = 0; i < dim_y; i++){
		hostMatrix[i] = (double*)malloc( sizeof(double) * dim_y);
		if (hostMatrix[i] == NULL){
			fprintf(stderr, "Error in Host Matrix allocation\n" );
			exit(EXIT_FAILURE);
		}
		for ( int j = 0; j < dim_y; j++) {
			hostMatrix[i][j] = j + (i * dim_x);
		}
	}

	deviceMatrix = NULL;
	hipMalloc((void**)&deviceMatrix, sizeof(double)*dim_x*dim_y);


	deviceMatrixTransposed = NULL;
	hipMalloc((void**)&deviceMatrixTransposed, sizeof(double)*dim_x*dim_y);




	//Kernel invocation with <<< # of blocks , # of threads per block>>>(args...)
	hello_from_GPU<<<10,10>>>();
	cudaCheckError();
	hipDeviceSynchronize();

	//Reset the GPU
	hipDeviceReset();

	return 0;
}

// Yey
